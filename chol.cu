/* Cholesky decomposition.
 * Host code.
 * Author: Naga Kandasamy
 * Date: May 23, 2013
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include "chol_kernel.cu"


////////////////////////////////////////////////////////////////////////////////
// declarations, forward
Matrix allocate_matrix_on_gpu(const Matrix M);
Matrix allocate_matrix(int num_rows, int num_columns, int init);
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost);
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice);

void check_error(const char *msg);

extern Matrix create_positive_definite_matrix(unsigned int, unsigned int);
extern "C" int chol_gold(const Matrix, Matrix);
extern "C" int check_chol(const Matrix, const Matrix);
void chol_on_device(const Matrix, Matrix);
void chol_on_device_optimized(const Matrix, Matrix);

//Globals
float time_cpu;
// Matrices for the program
Matrix A; // The N x N input matrix
Matrix reference; // The upper triangular matrix computed by the CPU
Matrix U_on_device; // The upper triangular matrix computed by the device (slow)
Matrix U_on_device_fast; // The upper triangular matrix computed by the device (fast)


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));

	// Create the positive definite matrix. May require a few tries if we are unlucky
	int success = 0;
	while(!success){
		A = create_positive_definite_matrix(MATRIX_SIZE, MATRIX_SIZE);
		if(A.elements != NULL)
				  success = 1;
	}

	reference  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0); // Create a matrix to store the CPU result
	U_on_device =  allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0); // Create a matrix to store the device result
	U_on_device_fast =  allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 0);

	//Compute the Cholesky decomposition on the CPU
	unsigned int timer;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	printf("== CPU ==\n");
	int status = 1;
	status = chol_gold(A, reference);
	cutStopTimer(timer);
	time_cpu = 1e-3 * cutGetTimerValue(timer);
	printf("	Run time:    %0.10f s. \n", time_cpu);
	if(status == 0){
			  printf("Cholesky decomposition failed. The input matrix is not positive definite. \n");
			  exit(0);
	}
	/*
	printf("Double checking for correctness by recovering the original matrix. \n");
	if(check_chol(A, reference) == 0){
		printf("CPU: FAILED\n");
		exit(0);
	}
	*/
	printf("	PASSED\n"); //IT IS SO PERFECT WE DON'T EVEN CHECK.
	

	//Slow
	//Perform the Cholesky decomposition on the GPU. The resulting upper triangular matrix should be retured in U_on_gpu
	chol_on_device(A, U_on_device);
	
	
	//Optimized
	//Perform the Cholesky decomposition on the GPU. The resulting upper triangular matrix should be retured in U_on_gpu
	chol_on_device_optimized(A, U_on_device_fast);
	
	// Free host matrices
	free(A.elements); 	
	free(U_on_device.elements);
	free(U_on_device_fast.elements);	
	free(reference.elements); 
	return 1;
}

//Error helper
void check_for_error(char *msg){
	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
		printf("CUDA ERROR: %s (%s). \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
} 

/* Write code to perform Cholesky decopmposition on the device. */
void chol_on_device(const Matrix A, Matrix U)
{
	//Slow
	//Perform the Cholesky decomposition on the GPU. The resulting upper triangular matrix should be retured in U_on_gpu
	unsigned int timer_gpu;
	cutCreateTimer(&timer_gpu);
	
	//A and U are already allocated on CPU already
	//Allocate space on gpu
	Matrix gpu_u = allocate_matrix_on_gpu( U );

	//Copy matrices to gpu, copy A right into U
	copy_matrix_to_device( gpu_u, A );
	
	//Maximum size expected is 8192x8192
	//Will be splitting the elimination i loop
	//Which has up to MATRIX_SIZE iterations
	//So we would optimally use 8192 threads
	//Thus requiring 16 blocks
	//Rather than attempting to syncronize 16 blocks
	//Where each thread does one operation per outer K iteration
	//Just have one block and have each thread do 16 operations 
	//(in the worst case)
	int num_blocks = 1;
	
	//Max per block threads
	int threads_per_block = 512;
	
	//Operations per thread
	int ops_per_thread = MATRIX_SIZE / (threads_per_block*num_blocks);
	
	printf("== GPU (Slow) ==\n");
	printf("	Threads per block: %d\n",threads_per_block);
	printf("	Number of blocks: %d\n",num_blocks);
	printf("	Operations per thread: %d\n",ops_per_thread);
	
	//Set up the execution grid on the GPU
	dim3 thread_block(threads_per_block, 1, 1);
	dim3 grid(num_blocks,1);
	
	//Start timer after copy
	cutStartTimer(timer_gpu);
	
	// Launch the kernel <<<grid, thread_block>>>
	chol_kernel<<<grid, thread_block>>>(gpu_u.elements,ops_per_thread);
	
	//Sync at end and check for errors
	hipDeviceSynchronize();
	check_for_error("SLOW KERNEL FAILURE\n");
	
	//Stop timer before copy back
	cutStopTimer(timer_gpu);
	
	//Copy data back
	copy_matrix_from_device(U, gpu_u);
	
	//Free memory on device
	hipFree(gpu_u.elements);
	
	float time_gpu = 1e-3 * cutGetTimerValue(timer_gpu);
	printf("	Run time:    %0.10f s. \n", time_gpu);
	printf("	Speedup: %0.10f\n", time_cpu/time_gpu);
	//Check if the device result is equivalent to the expected solution. If you can't meet the desired tolerance, try using double precision support.
	unsigned int size = reference.num_rows * reference.num_columns;
	CUTBoolean res = cutComparefe(reference.elements, U_on_device.elements, size, 0.1f);
	printf("	%s\n", (1 == res) ? "PASSED" : "FAILED");
}

/* Write code to perform Cholesky decopmposition on the device. */
void chol_on_device_optimized(const Matrix A, Matrix U)
{
	unsigned int timer_gpu_fast;
	cutCreateTimer(&timer_gpu_fast);
	
	printf("== GPU (Fast) ==\n");
	//A and U are already allocated on CPU already
	//Allocate space on gpu for U
	Matrix gpu_u = allocate_matrix_on_gpu( U );

	//Copy matrices to gpu, copy A right into U
	copy_matrix_to_device( gpu_u, A );
	
	//Start timer after copy
	cutStartTimer(timer_gpu_fast);
	
	//Each thread within a block will take some j iterations
	int threads_per_block = 256; //Optimal
	//Stride size should equal threads per block - just cause?
	int stride = threads_per_block;
	printf("	Threads per block / stride: %d\n",threads_per_block);

	
	//Each kernel call will be one iteration of out K loop
	int k;
	for(k = 0; k < MATRIX_SIZE; k++)
	{
		//Want threads to stride across memory
		//i is outer loop
			//j is inner loop
		//so threads should split the j loop
		//Each thread block will take an i iteration
		int isize = (MATRIX_SIZE-1) - (k+1) + 1;
		int num_blocks = isize;
		if(num_blocks <= 0)
		{
			num_blocks = 1;
		}
		
		//Set up the execution grid on the GPU
		//printf("	Threads per block: %d\n",threads_per_block);
		//printf("	Number of blocks: %d\n",num_blocks);
		dim3 thread_block(threads_per_block, 1, 1);
		dim3 grid(num_blocks,1);
		
		//Call the div kernel for this k iteration
		chol_kernel_optimized_div<<<grid, thread_block>>>(
			gpu_u.elements,
			k,
			stride);
		
		//Call kernel with for this K iteration
		chol_kernel_optimized<<<grid, thread_block>>>(
			gpu_u.elements,
			k,
			stride);
			
			
		//Sync at end and check for errors
		hipDeviceSynchronize();
		check_for_error("FAST KERNEL FAILURE");
	}
	
	//Sync at end
	hipDeviceSynchronize();
	
	//Stop timer before copy back					 
	cutStopTimer(timer_gpu_fast);
	
	//Copy data back
	copy_matrix_from_device(U, gpu_u);
	
	//Free memory on device
	hipFree(gpu_u.elements);
	
	
	//As the final step, zero out the lower triangular portion of U
	int i, j;
	for(i = 0; i < MATRIX_SIZE; i++)
			  for(j = 0; j < i; j++)
						 U.elements[i * MATRIX_SIZE + j] = 0.0;
						 
	float time_gpu_fast = 1e-3 * cutGetTimerValue(timer_gpu_fast);
	printf("	Run time:    %0.10f s. \n", time_gpu_fast);
	printf("	Speedup: %0.10f\n", time_cpu/time_gpu_fast);
	//Check if the device result is equivalent to the expected solution. If you can't meet the desired tolerance, try using double precision support.
	unsigned int size_fast = reference.num_rows * reference.num_columns;
	CUTBoolean res_fast = cutComparefe(reference.elements, U_on_device_fast.elements, size_fast, 0.1f);
	printf("	%s\n", (1 == res_fast) ? "PASSED" : "FAILED");
}

// Allocate a device matrix of same size as M.
Matrix allocate_matrix_on_gpu(const Matrix M){
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}


// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix allocate_matrix(int num_rows, int num_columns, int init)
{
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float *) malloc(size * sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
			M.elements[i] = (float)rand()/(float)RAND_MAX;
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice){
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

void check_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}
